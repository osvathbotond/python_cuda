#include "hip/hip_runtime.h"
#include <math.h>

#include "vector_functions.cuh"
#include "exceptions.hpp"


static const int num_threads_per_block = 512;


__global__ void sinKernel(const float* vec, float* res, const size_t vector_length) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < vector_length) {
        res[tid] = sinf(vec[tid]);
    }
}

void sinCuda(const float* d_vec, float* d_res, const size_t vector_length) {
    // ceil(vector_length / num_threads_per_block)
    int num_blocks = (vector_length + num_threads_per_block - 1) / num_threads_per_block;

    sinKernel<<<num_blocks, num_threads_per_block>>>(d_vec, d_res, vector_length);
    hipError_t err = hipSuccess;
    err = hipGetLastError();
    if (err != hipSuccess) {
        throw CudaKernelError(hipGetErrorString(err));
    }
}
