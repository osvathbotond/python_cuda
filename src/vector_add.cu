#include "hip/hip_runtime.h"
#include "vector_functions.cuh"
#include "exceptions.hpp"


static const int num_threads_per_block = 512;


__global__ void addKernel(const float* vec1, const float* vec2, float* res, const size_t vector_length) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < vector_length) {
        res[tid] = vec1[tid] + vec2[tid];
    }
}

void addCuda(const float* d_vec1, const float* d_vec2, float* d_res, const size_t vector_length) {
    // ceil(vector_length / num_threads_per_block)
    int num_blocks = (vector_length + num_threads_per_block - 1) / num_threads_per_block;

    addKernel<<<num_blocks, num_threads_per_block>>>(d_vec1, d_vec2, d_res, vector_length);
    hipError_t err = hipSuccess;
    err = hipGetLastError();
    if (err != hipSuccess) {
        throw CudaKernelError(hipGetErrorString(err));
    }
}
