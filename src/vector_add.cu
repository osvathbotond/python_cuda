#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <stdio.h>
#include <vector>
#include <cmath>
#include <iostream>
#include <fstream>
#include <random>
#include <typeinfo>
#include <thread>
#include <cstdlib>
#include <algorithm>
#include <cassert>
#include <numeric>
#include <string>

#include "vector_functions.cuh"
#include "exceptions.hpp"


static const int num_threads = 512;

__global__ void addKernel(const float* vec1, const float* vec2, float* res, size_t vector_length) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < vector_length) {
        res[tid] = vec1[tid] + vec2[tid];
    }
}

void add(float* vec1, float* vec2, float* res, const size_t vector_length) {
    size_t bytes = vector_length * sizeof(float);

    // ceil(vector_length / num_threads)
    int num_blocks = (vector_length + num_threads - 1) / num_threads;

    // Pointers to the device-side variables
    float *d_vec1, *d_vec2, *d_res;

    // Allocate the memory on the GPU and move the vector (with error handling)
    hipError_t err = hipSuccess;
    err = hipMalloc(&d_vec1, bytes);
    if (err != hipSuccess) {
        throw CudaMallocError(hipGetErrorString(err));
    }

    err = hipMalloc(&d_vec2, bytes);
    if (err != hipSuccess) {
        throw CudaMallocError(hipGetErrorString(err));
    }

    err = hipMalloc(&d_res, bytes);
    if (err != hipSuccess) {
        throw CudaMallocError(hipGetErrorString(err));
    }

    err = hipMemcpy(d_vec1, vec1, bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        throw CudaCopyError(hipGetErrorString(err));
    }

    err = hipMemcpy(d_vec2, vec2, bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        throw CudaCopyError(hipGetErrorString(err));
    }

    // const float* vec1, const float* vec2, float* res, size_t vector_length
    addKernel<<<num_blocks, num_threads>>>(d_vec1, d_vec2, d_res, vector_length);
    err = hipGetLastError();
    if (err != hipSuccess) {
        throw CudaKernelError(hipGetErrorString(err));
    }

    // Copying back to the host
    err = hipMemcpy(res, d_res, bytes, hipMemcpyDeviceToHost);

    if (err != hipSuccess) {
        throw CudaCopyError(hipGetErrorString(err));
    }

    // Freeing the memory on the device. Not doing so can cause memory-leak.
    err = hipFree(d_vec1);
    if (err != hipSuccess) {
        throw CudaFreeError(hipGetErrorString(err));
    }

    err = hipFree(d_vec2);
    if (err != hipSuccess) {
        throw CudaFreeError(hipGetErrorString(err));
    }

    err = hipFree(d_res);
    if (err != hipSuccess) {
        throw CudaFreeError(hipGetErrorString(err));
    }
}
