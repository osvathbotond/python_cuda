#include "hip/hip_runtime.h"
#include "vector_functions.cuh"
#include "exceptions.hpp"


static const int num_threads = 512;


__global__ void scaleKernel(const float* vec, const float scalar, float* res, const size_t vector_length) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < vector_length) {
        res[tid] = vec[tid] * scalar;
    }
}

void scaleCuda(const float* d_vec, const float scalar, float* d_res, const size_t vector_length) {
    // ceil(vector_length / num_threads)
    int num_blocks = (vector_length + num_threads - 1) / num_threads;

    scaleKernel<<<num_blocks, num_threads>>>(d_vec, scalar, d_res, vector_length);
    hipError_t err = hipSuccess;
    err = hipGetLastError();
    if (err != hipSuccess) {
        throw CudaKernelError(hipGetErrorString(err));
    }
}


