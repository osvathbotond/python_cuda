#include "memory.cuh"
#include "exceptions.hpp"


void* _cudaMalloc(size_t mySize) {
    void* ptr;
    hipError_t err = hipSuccess;
    hipMalloc((void**)&ptr, mySize);
    if (err != hipSuccess) {
        throw CudaMallocError(hipGetErrorString(err));
    }
    return ptr;
}

void cudaFreeDeleter::operator()(float* ptr) const {
    hipError_t err = hipSuccess;
    hipFree(ptr);
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }
}

cuda_shared_ptr::cuda_shared_ptr(float* ptr) : std::shared_ptr<float>(ptr, cudaFreeDeleter()) {}

float* cuda_shared_ptr::get() const {
    return std::shared_ptr<float>::get();
}

float* cuda_shared_ptr::release() {
    float* releasedPtr = std::shared_ptr<float>::get();
    reset();
    return releasedPtr;
}
cuda_shared_ptr getDevicePointer(size_t bytes){

    cuda_shared_ptr d_data((float*)_cudaMalloc(bytes));

    return d_data;
}

void cudaDevice2Host(float* data, cuda_shared_ptr d_data, size_t bytes) {
    hipError_t err = hipSuccess;
    err = hipMemcpy(data, d_data.get(), bytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        throw CudaCopyError(hipGetErrorString(err));
    }
}

void cudaHost2Devide(cuda_shared_ptr d_data, float* data, size_t bytes) {
    hipError_t err = hipSuccess;
    err = hipMemcpy(d_data.get(), data, bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        throw CudaCopyError(hipGetErrorString(err));
    }
}
